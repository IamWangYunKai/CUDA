
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define NUM 1024

hipError_t addWithCuda(double *c, const double *a, const double *b, long size);
__device__ double test(int i);

__global__ void addKernel(double *c, const double *a, const double *b)
{
	int i = threadIdx.x;
	for (int j = 0; j < NUM; j++) {
		c[j] = test(i);
	}
}
__device__ double test(int i) {
	double tmp = (i - 9.9) * 3.1415926 * 3.1415926;
	double ret = sin(tmp);
	return ret;
}

int main()
{
    const double a[NUM] = { 1, 2, 3, 4, 5 };
    const double b[NUM] = { 10, 20, 30, 40, 50 };
	double c[NUM] = { 0 };

    // Add vectors in parallel.
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
    hipError_t cudaStatus = addWithCuda(c, a, b, NUM);
	hipEventRecord(stop, 0);
	//confirm that all things have been done before "stop event"
	hipEventSynchronize(stop);
	float elapseTime;
	hipEventElapsedTime(&elapseTime, start, stop);
	printf("Time for I/O : %.5f ms\n", elapseTime);
	printf("%.6f\n", c[2]);

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }
	system("pause");
    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(double *c, const double *a, const double *b, long size)
{
	double *dev_a = 0;
	double *dev_b = 0;
	double *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
