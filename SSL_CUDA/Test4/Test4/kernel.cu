#include "hip/hip_runtime.h"
#include <stdio.h> 
#include "hip/hip_runtime.h"
#include ""
#include "kernel.h"  
#include "CU_geometry.h"

__global__ void kernel()
{
	printf("hello world!\n");
	CGeoPoint point = CGeoPoint(450, 0);
	printf("%.2f, %.2f\n", point.x(), point.y());
}

void test() {
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	kernel << <1, 1 >> >();

	hipEventRecord(stop, 0);
	//confirm that all things have been done before "stop event"
	hipEventSynchronize(stop);
	float elapseTime;
	hipEventElapsedTime(&elapseTime, start, stop);
	printf("Time for I/O : %.5f ms\n", elapseTime);
}