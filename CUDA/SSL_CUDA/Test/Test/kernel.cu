#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
//void test(const double *ourplay_x, const double *ourplay_y, const double *ourplay_dir, const double *ourplay_vx, const double *ourplay_vy, const double *ourplay_vr, unsigned int size);

void test(const double *ourplay_x, const double *ourplay_y, const double *ourplay_dir, const double *ourplay_vx, const double *ourplay_vy, const double *ourplay_vr, unsigned int size) {
	double *dev_ourplay_x;
	double *dev_ourplay_y;
	double *dev_ourplay_dir;
	double *dev_ourplay_vx;
	double *dev_ourplay_vy;
	double *dev_ourplay_vr;
	hipSetDevice(0);
	hipMalloc((void**)&dev_ourplay_x, size * sizeof(double));
	hipMalloc((void**)&dev_ourplay_y, size * sizeof(double));
	hipMalloc((void**)&dev_ourplay_dir, size * sizeof(double));
	hipMalloc((void**)&dev_ourplay_vx, size * sizeof(double));
	hipMalloc((void**)&dev_ourplay_vy, size * sizeof(double));
	hipMalloc((void**)&dev_ourplay_vr, size * sizeof(double));
	hipMemcpy(dev_ourplay_x, ourplay_x, size * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_ourplay_y, ourplay_y, size * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_ourplay_dir, ourplay_dir, size * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_ourplay_vx, ourplay_vx, size * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_ourplay_vy, ourplay_vy, size * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_ourplay_vr, ourplay_vr, size * sizeof(double), hipMemcpyHostToDevice);
	//do nothing
	hipFree(dev_ourplay_x);
	hipFree(dev_ourplay_y);
	hipFree(dev_ourplay_dir);
	hipFree(dev_ourplay_vx);
	hipFree(dev_ourplay_vy);
	hipFree(dev_ourplay_vr);
}

void main() {
	const unsigned int num = 12;
	double ourplay_x[num] = { 0 };
	double ourplay_y[num] = { 0 };
	double ourplay_dir[num] = { 0 };
	double ourplay_vx[num] = { 0 };
	double ourplay_vy[num] = { 0 };
	double ourplay_vr[num] = { 0 };
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	test(ourplay_x, ourplay_y, ourplay_dir, ourplay_vx, ourplay_vy, ourplay_vr, num);
	hipEventRecord(stop, 0);
	//confirm that all things have been done before "stop event"
	hipEventSynchronize(stop);
	float elapseTime;
	hipEventElapsedTime(&elapseTime, start, stop);
	printf("Time for I/O : %.5f ms\n", elapseTime);
	system("pause");
}