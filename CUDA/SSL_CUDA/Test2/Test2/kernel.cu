#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
//void test(const double *ourplay_x, const double *ourplay_y, const double *ourplay_dir, const double *ourplay_vx, const double *ourplay_vy, const double *ourplay_vr, unsigned int size);
const unsigned int NUM = 12;
__constant__ double dev_ourplay_x[NUM];
__constant__ double dev_ourplay_y[NUM];
__constant__ double dev_ourplay_dir[NUM];
__constant__ double dev_ourplay_vx[NUM];
__constant__ double dev_ourplay_vy[NUM];
__constant__ double dev_ourplay_vr[NUM];

void test(const double *ourplay_x, const double *ourplay_y, const double *ourplay_dir, const double *ourplay_vx, const double *ourplay_vy, const double *ourplay_vr, unsigned int size) {
	/*
	hipMalloc((void**)&dev_ourplay_x, size * sizeof(double));
	hipMalloc((void**)&dev_ourplay_y, size * sizeof(double));
	hipMalloc((void**)&dev_ourplay_dir, size * sizeof(double));
	hipMalloc((void**)&dev_ourplay_vx, size * sizeof(double));
	hipMalloc((void**)&dev_ourplay_vy, size * sizeof(double));
	hipMalloc((void**)&dev_ourplay_vr, size * sizeof(double));
	*/
	hipMemcpyToSymbol(HIP_SYMBOL(dev_ourplay_x), ourplay_x, size * sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(dev_ourplay_y), ourplay_y, size * sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(dev_ourplay_dir), ourplay_dir, size * sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(dev_ourplay_vx), ourplay_vx, size * sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(dev_ourplay_vy), ourplay_vy, size * sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(dev_ourplay_vr), ourplay_vr, size * sizeof(double));
	//do nothing
	/*
	hipFree(dev_ourplay_x);
	hipFree(dev_ourplay_y);
	hipFree(dev_ourplay_dir);
	hipFree(dev_ourplay_vx);
	hipFree(dev_ourplay_vy);
	hipFree(dev_ourplay_vr);
	*/
}

void main() {
	const unsigned int num = 12;
	double ourplay_x[num] = { 0 };
	double ourplay_y[num] = { 0 };
	double ourplay_dir[num] = { 0 };
	double ourplay_vx[num] = { 0 };
	double ourplay_vy[num] = { 0 };
	double ourplay_vr[num] = { 0 };
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	test(ourplay_x, ourplay_y, ourplay_dir, ourplay_vx, ourplay_vy, ourplay_vr, num);
	hipEventRecord(stop, 0);
	//confirm that all things have been done before "stop event"
	hipEventSynchronize(stop);
	float elapseTime;
	hipEventElapsedTime(&elapseTime, start, stop);
	printf("Time for I/O : %.5f ms\n", elapseTime);
	system("pause");
}